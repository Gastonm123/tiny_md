#include "hip/hip_runtime.h"
#include "core.hh"
#include "parameters.h"

#include <math.h>
#include <stdlib.h> // rand()
#include <stdio.h>
#include <omp.h>

#define DIV_CEIL(a,b) (((a)+(b)-1)/(b))
#define ECUT (4.0f * (powf(RCUT, -12) - powf(RCUT, -6)))
#define X_OFF 0
#define Y_OFF N
#define Z_OFF (2*N)

static inline int myrand(int *state) {
    int x = *state;
    x ^= x << 13;
    x ^= x >> 17;
    x ^= x << 5;
    *state = x;
    return x;
}

void init_pos(float* rxyz, const float rho)
{
    // inicialización de las posiciones de los átomos en un cristal FCC

    float a = cbrtf(4.0f / rho);
    int nucells = ceilf(cbrtf((float)N / 4.0f));
    int idx = 0;

    for (int i = 0; i < nucells; i++) {
        for (int j = 0; j < nucells; j++) {
            for (int k = 0; k < nucells; k++) {
                // se pueden tomar de a cuatro puntos en tres vectores
                // y con una multiplicacion se termina

                rxyz[X_OFF + idx + 0] = i * a; // x
                rxyz[Y_OFF + idx + 0] = j * a; // y
                rxyz[Z_OFF + idx + 0] = k * a; // z
                    // del mismo átomo
                rxyz[X_OFF + idx + 1] = (i + 0.5f) * a;
                rxyz[Y_OFF + idx + 1] = (j + 0.5f) * a;
                rxyz[Z_OFF + idx + 1] = k * a;

                rxyz[X_OFF + idx + 2] = (i + 0.5f) * a;
                rxyz[Y_OFF + idx + 2] = j * a;
                rxyz[Z_OFF + idx + 2] = (k + 0.5f) * a;

                rxyz[X_OFF + idx + 3] = i * a;
                rxyz[Y_OFF + idx + 3] = (j + 0.5f) * a;
                rxyz[Z_OFF + idx + 3] = (k + 0.5f) * a;

                idx += 4;
            }
        }
    }
}

void init_vel(float* vxyz, float* temp, float* ekin)
{
    // inicialización de velocidades aleatorias

    float sf, sumvx = 0.0f, sumvy = 0.0f, sumvz = 0.0f, sumv2 = 0.0f;
    int state = SEED;

    for (int i = 0; i < N; ++i) {
        vxyz[X_OFF + i] = myrand(&state) / (float)RAND_MAX - 0.5f;
        vxyz[Y_OFF + i] = myrand(&state) / (float)RAND_MAX - 0.5f;
        vxyz[Z_OFF + i] = myrand(&state) / (float)RAND_MAX - 0.5f;

        sumvx += vxyz[X_OFF + i];
        sumvy += vxyz[Y_OFF + i];
        sumvz += vxyz[Z_OFF + i];
        sumv2 += vxyz[X_OFF + i] * vxyz[X_OFF + i] + vxyz[Y_OFF + i] * vxyz[Y_OFF + i]
            + vxyz[Z_OFF + i] * vxyz[Z_OFF + i];
    }

    sumvx /= (float)N;
    sumvy /= (float)N;
    sumvz /= (float)N;
    *temp = sumv2 / (3.0f * N);
    *ekin = 0.5f * sumv2;
    sf = sqrtf(T0 / *temp);

    for (int i = 0; i < N; ++i) { // elimina la velocidad del centro de masa
        // y ajusta la temperatura
        vxyz[X_OFF + i] = (vxyz[X_OFF + i] - sumvx) * sf;
        vxyz[Y_OFF + i] = (vxyz[Y_OFF + i] - sumvy) * sf;
        vxyz[Z_OFF + i] = (vxyz[Z_OFF + i] - sumvz) * sf;
    }
}


__device__ float minimum_image(float cordi, const float cell_length)
{
    // imagen más cercana

    if (cordi <= -0.5f * cell_length) {
        cordi += cell_length;
    } else if (cordi > 0.5f * cell_length) {
        cordi -= cell_length;
    }
    return cordi;
}

__global__ void forces_naive(const float *rxyz, float *fxyz, float *epot, float *pres,
                             const float L) 
{
	// int tid = threadIdx.x;
	// int lane = tid % warpSize;
	int gtid = blockIdx.x * blockDim.x + threadIdx.x;

    if (gtid >= N)
        return;

    const float rcut2 = RCUT * RCUT;
    float xi = rxyz[X_OFF + gtid];
    float yi = rxyz[Y_OFF + gtid];
    float zi = rxyz[Z_OFF + gtid];

    for (int j = 0 ; j < N; j++) {
        if (j == gtid) continue;

        float xj = rxyz[X_OFF + j];
        float yj = rxyz[Y_OFF + j];
        float zj = rxyz[Z_OFF + j];

        // distancia mínima entre r_i y r_j
        float rx = xi - xj;
        rx = minimum_image(rx, L);
        float ry = yi - yj;
        ry = minimum_image(ry, L);
        float rz = zi - zj;
        rz = minimum_image(rz, L);

        float rij2 = rx * rx + ry * ry + rz * rz;

        if (rij2 <= rcut2) {
            float r2inv = 1.0f / rij2;
            float r6inv = r2inv * r2inv * r2inv;

            float fr = 24.0f * r2inv * r6inv * (2.0f * r6inv - 1.0f);

            fxyz[X_OFF + gtid] += fr * rx;
            fxyz[Y_OFF + gtid] += fr * ry;
            fxyz[Z_OFF + gtid] += fr * rz;

            atomicAdd(epot, 4.0f * r6inv * (r6inv - 1.0f) - ECUT);
            atomicAdd(pres, fr * rij2);
        }
    }
}

void forces(const float* rxyz, float* fxyz, float* epot, float* pres,
            const float* temp, const float rho, const float V, const float L)
{
    // calcula las fuerzas LJ (12-6)

    for (int i = 0; i < 3 * N; i+=3) {
        fxyz[i + 0] = 0.0f;
        fxyz[i + 1] = 0.0f;
        fxyz[i + 2] = 0.0f;
    }
    float pres_vir;

    const int BLOCK_SIZE = 1024;
    
    float *d_rxyz = NULL, *d_fxyz = NULL, *d_epot = NULL, *d_pres = NULL;
    const int ARRAY_SIZE = 3 * N * sizeof(float);
    hipMalloc(&d_rxyz, ARRAY_SIZE);
    hipMalloc(&d_fxyz, ARRAY_SIZE);
    hipMalloc(&d_epot, sizeof(float));
    hipMalloc(&d_pres, sizeof(float));

    hipMemcpy(d_rxyz, rxyz, ARRAY_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_fxyz, fxyz, ARRAY_SIZE, hipMemcpyHostToDevice);

    forces_naive<<<BLOCK_SIZE,DIV_CEIL(N,BLOCK_SIZE)>>>(d_rxyz, d_fxyz, d_epot, d_pres, L);
    hipDeviceSynchronize();

    hipMemcpy(fxyz, d_fxyz, ARRAY_SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(epot, d_epot, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&pres_vir, d_pres, sizeof(float), hipMemcpyDeviceToHost);

    // presion y energia potencial se cuentan dos veces
    
    *epot /= 2.0f;
    pres_vir /= (V * 6.0f);
    *pres = *temp * rho + pres_vir;
}


static float pbc(float cordi, const float cell_length)
{
    // condiciones periodicas de contorno coordenadas entre [0,L)
    if (cordi <= 0.0f) {
        cordi += cell_length;
    } else if (cordi > cell_length) {
        cordi -= cell_length;
    }
    return cordi;
}


void velocity_verlet(float* rxyz, float* vxyz, float* fxyz, float* epot,
                     float* ekin, float* pres, float* temp, const float rho,
                     const float V, const float L)
{

    for (int i = 0; i < N; ++i) { // actualizo posiciones
        rxyz[X_OFF + i] += vxyz[X_OFF + i] * DT + 0.5f * fxyz[X_OFF + i] * DT * DT;
        rxyz[Y_OFF + i] += vxyz[Y_OFF + i] * DT + 0.5f * fxyz[Y_OFF + i] * DT * DT;
        rxyz[Z_OFF + i] += vxyz[Z_OFF + i] * DT + 0.5f * fxyz[Z_OFF + i] * DT * DT;

        rxyz[X_OFF + i] = pbc(rxyz[X_OFF + i], L);
        rxyz[Y_OFF + i] = pbc(rxyz[Y_OFF + i], L);
        rxyz[Z_OFF + i] = pbc(rxyz[Z_OFF + i], L);

        vxyz[X_OFF + i] += 0.5f * fxyz[X_OFF + i] * DT;
        vxyz[Y_OFF + i] += 0.5f * fxyz[Y_OFF + i] * DT;
        vxyz[Z_OFF + i] += 0.5f * fxyz[Z_OFF + i] * DT;
    }

    forces(rxyz, fxyz, epot, pres, temp, rho, V, L); // actualizo fuerzas

    float sumv2 = 0.0f;
    for (int i = 0; i < N; ++i) { // actualizo velocidades
        vxyz[X_OFF + i] += 0.5f * fxyz[X_OFF + i] * DT;
        vxyz[Y_OFF + i] += 0.5f * fxyz[Y_OFF + i] * DT;
        vxyz[Z_OFF + i] += 0.5f * fxyz[Z_OFF + i] * DT;

        sumv2 += vxyz[X_OFF + i] * vxyz[X_OFF + i] + vxyz[Y_OFF + i] * vxyz[Y_OFF + i]
            + vxyz[Z_OFF + i] * vxyz[Z_OFF + i];
    }

    *ekin = 0.5f * sumv2;
    *temp = sumv2 / (3.0f * N);
}
